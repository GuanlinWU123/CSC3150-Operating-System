﻿#include "file_system.h"
#include <cstdint>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ __managed__ u32 gtime = 0;

__device__ void make_root_directory(FileSystem* fs) {
  FCB root(fs, 0, gtime++, "root");
  root.set_to_directory();
  root.set_current_dir(0);
}

__device__ FCB get_cd(FileSystem* fs) {
  FCB root(fs, 0);
  return FCB(fs, root.get_current_dir());
}

__device__ void set_cd(FileSystem* fs, uint16_t index) {
  FCB root(fs, 0);
  root.set_current_dir(index);
}

__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
                        int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
                        int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE,
                        int MAX_FILE_NUM, int MAX_FILE_SIZE,
                        int FILE_BASE_ADDRESS) {
  // init variables
  fs->volume = volume;

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;
  make_root_directory(fs);
}

__device__ u32 fs_open(FileSystem *fs, char *s, int op) {
  /* Implement open operation here */
  FCB first_unused_entry;

  for (u32 i = 0; i < fs->MAX_FILE_NUM; i++) {

    FCB current(fs, i);

    if (current.is_unused() && first_unused_entry.is_null()) {
      first_unused_entry = current;
    } else if (mystrcmp(s, current.get_filename()) && current.get_parent_index() == get_cd(fs).index) {
      current.set_modified_timestamp(gtime++);
      return i;
    }
  }

  if (first_unused_entry.is_null()) {
    printf("storage is full.\n");
    return 0xFFFFFFFF;
  }
  first_unused_entry.set_filename(s);
  first_unused_entry.set_file_size(0);
  uint16_t current_time = gtime++;
  first_unused_entry.set_creation_timestamp(current_time);
  first_unused_entry.set_modified_timestamp(current_time);

  get_cd(fs).add_file_to_directory(first_unused_entry);
  return first_unused_entry.index;
}

__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp) {
  /* Implement read operation here */
  if (fp > fs->MAX_FILE_NUM) {
    return;
  }
  FCB file_to_read(fs, fp);
  size = (file_to_read.get_file_size() < size) ? file_to_read.get_file_size() : size;

  uchar *start = file_to_read.get_start_content_ptr();
  for (u32 i = 0; i < size; i++) {
    output[i] = start[i];
  }
}

__device__ u32 fs_write(FileSystem *fs, uchar *input, u32 size, u32 fp) {
  /* Implement write operation here */
  if (fp > fs->MAX_FILE_NUM) {
    return 0xFFFFFFFF;
  }

  FCB file(fs, fp);
  SuperBlock sb(fs);
  if (size > fs->MAX_FILE_SIZE / fs->MAX_FILE_NUM) {
    return 0xFFFFFFFF;
  }

  FCB parent_dir = file.get_parent();
  file.set_file_size(size);

  uint16_t current_time = gtime++;
  file.set_modified_timestamp(current_time);

  for (u32 i = file.get_start_block_index(); i < file.get_ending_block_index();
       i++) {
    sb.set_used(i);
  }

  uchar *start = file.get_start_content_ptr();
  for (u32 i = 0; i < size; i++) {
    start[i] = input[i];
  }
  return 0;
}

__device__ void sorting(FCB files_to_sort[], u32 size, int op)
{
  u32 greatest = 0;
  for (u32 i = 0; i < size; i++) {
    greatest = i;
    for (u32 j = i + 1; j < size; j++) {
      FCB current = files_to_sort[greatest];
      FCB next = files_to_sort[j];
      if (op == LS_D) {
        if (next.get_modified_timestamp() > current.get_modified_timestamp()) {
          greatest = j;
        }
      } else if (op == LS_S) {
        if (next.get_file_size() > current.get_file_size()) {
          greatest = j;
        } else if (next.get_file_size() == current.get_file_size() && next.get_creation_timestamp() < current.get_creation_timestamp()) {
          greatest = j;
        }
      }
    }

    if (greatest != i) {
      FCB tmp = files_to_sort[i];
      files_to_sort[i] = files_to_sort[greatest];
      files_to_sort[greatest] = tmp;
    }
  }
}

__device__ FCB find_fcb(FileSystem *fs, const char* name) {
  FCB current = get_cd(fs).get_next_sub_file();
  while (current.index != 0) {
    if (mystrcmp(current.get_filename(), name)) {
        return current;
      }
      current = current.get_next_sibiling();
  }
  return FCB();
}

__device__ void print_pwd(FCB current) {
  if (!current.is_root_dir()) {
    print_pwd(current.get_parent());
    printf("%s/", current.get_filename());
  } else {
    printf("/");
  }
}

__device__ void fs_gsys(FileSystem *fs, int op) {
  /* Implement LS_D and LS_S operation here */
  if (op == CD_P) {
    set_cd(fs, get_cd(fs).get_parent_index());
    return;
  } else if (op == PWD) {
    FCB cd = get_cd(fs);
    print_pwd(cd);
    printf("\n");
    return;
  } else {
    uint32_t size = 0;
    FCB* files_to_sort = NULL;
    FCB cd = get_cd(fs);
    hipMalloc(&files_to_sort, fs->MAX_FILE_NUM*sizeof(FCB));
    FCB current = cd.get_next_sub_file();

    while (current.index != 0 && !current.is_unused()) {
      files_to_sort[size++] = current;
      current = current.get_next_sibiling();
    }
    sorting(files_to_sort, size, op);
    if (op == LS_D) {
      printf("=== sort by modification time ===\n");
      for (uint32_t i = 0; i < size; i++) {
        FCB& c = files_to_sort[i];
        printf("%s %c\n", c.get_filename(), c.is_directory() ? 'd' : ' ');
      }
    } else if (op == LS_S) {
      printf("=== sort by size ===\n");
      for (uint32_t i = 0; i < size; i++) {
        FCB& c = files_to_sort[i];
        printf("%s %d %c\n", c.get_filename(), c.get_file_size(), c.is_directory() ? 'd' : ' ');
      }
    }
    return;
  }
}

__device__ void fs_gsys(FileSystem *fs, int op, char *s) {
  /* Implement rm operation here */
  SuperBlock sb(fs);
  FCB current_dir = get_cd(fs);

  if (op == RM) {
    FCB file = find_fcb(fs, s);
    if (file.is_file()) {
      for (u32 i = file.get_start_block_index(); i < file.get_ending_block_index(); i++) {
        sb.set_free(i);
      }
      file.set_file_size(0);
      file.clean_filename();
    }
    return;
  } else if (op == MKDIR) {
    for (u32 i = 0; i < fs->MAX_FILE_NUM; i++) {
      FCB current(fs, i);
      if (current.is_unused()) {
        current = FCB(fs, i, gtime++, s);
        current.set_to_directory();
        current_dir.add_file_to_directory(current);
        break;
      }
    }
    return;
  } else if (op == CD) {
    FCB dir = find_fcb(fs, s);
    if (!dir.is_null()) {
      set_cd(fs, dir.index);
    } else {
      printf("dir not found!\n");
    }
    return;
  } else if (op == RM_RF) {
    FCB dir = find_fcb(fs, s);
    dir.remove_directory();
    return;
  }
}
